#include "hip/hip_runtime.h"
﻿
#include<iostream>
#include "hip/hip_runtime.h"
#include<string>
#include<vector>
#include<fstream>
#include <hip/hip_runtime.h>
#include ""
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>


# define PI  3.14159265358979323846

static void HandleError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		std::cout << hipGetErrorString(err) << "in" << file << "at line" << line;
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ )) 



__global__ void Eout_Ein_calculation(float* Eo_gpu, float* Ei_gpu, float* fo_gpu, float* fi_gpu, float* input_img_gpu, float* input, float* fout, float* fin, int img_w, int img_h,  int sigma) {

	
	int i = blockDim.y * blockIdx.y + threadIdx.y;	// calculate row index, point to the output  //width 
	int j = blockDim.x * blockIdx.x + threadIdx.x;	// calculate column index, point to the output //height

	float s1 = 0;
	float s2 = 0;

	for (int u = -sigma; u <= sigma; u++)
	{
		for (int v = -sigma; v <= sigma; v++)
		{
			if (((i + u) >= 0) && ((i + u) < img_h) && ((j + v) >= 0) && ((j + v) < img_w))
			{
				s1 = (s1 + ((1 / pow(((2 * sigma) + 1), 2)) * (input_img_gpu[((i + u) * img_w) + (j + v)] - fo_gpu[(i * img_w) + j]) * (input_img_gpu[((i + u) * img_w) + (j + v)] - fo_gpu[(i * img_w) + j])));
				s2 = (s2 + ((1 / pow(((2 * sigma) + 1), 2)) * (input_img_gpu[((i + u) * img_w) + (j + v)] - fi_gpu[(i * img_w) + j]) * (input_img_gpu[((i + u) * img_w) + (j + v)] - fi_gpu[(i * img_w) + j])));
			}
		}
	}
	Eo_gpu[i * img_w + j] = s1;
	Ei_gpu[i * img_w + j] = s2;

	

}

// convolution on device
__global__ void Convolution__on_device(float* out, float* img, float* kernel, int img_w,  int out_w, int out_h, int K) {
	size_t i = blockDim.y * blockIdx.y + threadIdx.y;
	size_t j = blockDim.x * blockIdx.x + threadIdx.x;

	// i and j being smaller than output's width and height, manage the edges perfectly
	if (i >= out_h || j >= out_w) return;

	float conv = 0;
	for (int ki = 0; ki < K; ki++)
		for (int kj = 0; kj < K; kj++)
			conv += img[(i + ki) * img_w + j + kj] * kernel[ki * K + kj];

	out[i * out_w + j] = conv;

}



void adddevice_convolution(float* y_output, float* in_img,  int img_w, int img_h, float sigma, float* gkernel , unsigned int k_size) {


	hipDeviceProp_t props;
	HANDLE_ERROR(hipGetDeviceProperties(&props, 0));


	// allocating the output array for pixels after convolution along y axis
	int y_height = img_h - k_size + 1;
	int y_width = img_w - k_size + 1;
	int y_size = y_height * y_width ;

	
	float* gkernel_gpu;
	float* input_img_gpu;
	float* gpu_output_y;
	size_t bytes = (img_w * img_h) * sizeof(float);
	

	HANDLE_ERROR(hipMalloc(&gkernel_gpu, k_size * k_size * sizeof(float)));
	HANDLE_ERROR(hipMalloc(&input_img_gpu, bytes));  							    //allocate memory on device
	HANDLE_ERROR(hipMalloc(&gpu_output_y, y_size * sizeof(float)));  				//allocate memory on device


	HANDLE_ERROR(hipMemcpy(input_img_gpu, in_img, bytes, hipMemcpyHostToDevice));     //copy the array from main memory to device
	HANDLE_ERROR(hipMemcpy(gkernel_gpu, gkernel, k_size * k_size * sizeof(float), hipMemcpyHostToDevice));     //copy the array from main memory to device


	size_t blockDim = sqrt(props.maxThreadsPerBlock);
	dim3 threads(blockDim, blockDim);
	dim3 blocks(img_w / threads.x +1, img_h / threads.y +1);

	Convolution__on_device << < blocks, threads >> > (gpu_output_y, input_img_gpu, gkernel_gpu, img_w, y_width, img_h, k_size);
	HANDLE_ERROR(hipMemcpy(y_output, gpu_output_y, y_size * sizeof(float), hipMemcpyDeviceToHost));

	
	
	hipFree(gpu_output_y);
	hipFree(gkernel_gpu);
	hipFree(input_img_gpu); 


}

void adddevice(float* input, float* fout, float* fin, float* Eo, float* Ei, int img_w, int img_h, int sigma) {
	
	
	hipDeviceProp_t props;
	HANDLE_ERROR(hipGetDeviceProperties(&props, 0));


	size_t blockDim = sqrt(props.maxThreadsPerBlock);
	dim3 threads(blockDim, blockDim);
	dim3 blocks(img_w / threads.x , img_h / threads.y );


	float* Eo_gpu;
	float* Ei_gpu;
	float* fo_gpu;
	float* fi_gpu;
	float* input_img_gpu;
	size_t bytes = (img_w * img_h) * sizeof(float);
	HANDLE_ERROR(hipMalloc(&input_img_gpu, bytes));  							    //allocate memory on device
	HANDLE_ERROR(hipMalloc(&fo_gpu,  bytes));  							//allocate memory on device
	HANDLE_ERROR(hipMalloc(&fi_gpu, bytes));  							//allocate memory on device
	HANDLE_ERROR(hipMalloc(&Eo_gpu, bytes));  							//allocate memory on device
	HANDLE_ERROR(hipMalloc(&Ei_gpu, bytes));  							//allocate memory on device

	HANDLE_ERROR(hipMemcpy(input_img_gpu, input, bytes, hipMemcpyHostToDevice));     //copy the array from main memory to device
	HANDLE_ERROR(hipMemcpy(fo_gpu, fout, bytes, hipMemcpyHostToDevice));     //copy the array from main memory to device
	HANDLE_ERROR(hipMemcpy(fi_gpu, fin, bytes, hipMemcpyHostToDevice));     //copy the array from main memory to device

	Eout_Ein_calculation << < blocks, threads >> > (Eo_gpu, Ei_gpu, fo_gpu, fi_gpu, input_img_gpu, input, fout, fin, img_w, img_h,  sigma);
	

	HANDLE_ERROR(hipMemcpy(Eo , Eo_gpu, bytes, hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(Ei , Ei_gpu, bytes, hipMemcpyDeviceToHost));


	hipFree(Eo_gpu);
	hipFree(Ei_gpu);
	hipFree(fo_gpu);
	hipFree(fi_gpu);
	hipFree(input_img_gpu);


}

